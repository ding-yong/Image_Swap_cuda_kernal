#include "hip/hip_runtime.h"
//kernel.cu
#include "hip/hip_runtime.h"
#include ""
#include <opencv2/highgui.hpp>
#include <stdio.h>
#include<math.h>
#include <iostream>
#include <sstream>
#include <opencv2/opencv.hpp>
#include <ctime>
#include "opencv2/core/cuda.hpp"

using namespace std;
using namespace cv;
using namespace cv::cuda;


//�Զ����ں˺���
__global__ void swap_rb_kernel(const PtrStepSz<uchar3> src1, const PtrStepSz<uchar3> src2, PtrStep<uchar3> dst, int left_x, int right_x)
{
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	int i = threadIdx.y + blockIdx.y * blockDim.y;

	int src1pix = src1(i, j).x + src1(i, j).y + src1(i, j).z;
	int src2pix = src2(i, j).x + src2(i, j).y + src2(i, j).z;

	if (i >= 0 && j >= 0)
	{
		//if (j < left_x)
		if (src2pix == 0)
		{
			dst(i, j) = src1(i, j);
		}
		else if (src1pix == 0)//(j >= right_x)
		{
			dst(i, j) = src2(i, j);
		}
		else
		{
			//float d = (j - left_x) / (float)X;
			float srcLen = abs(j - left_x);
			float warpLen = abs(j - right_x);
			float d = srcLen / (srcLen + warpLen);

			dst(i, j).x = (uchar)(src1(i, j).x * (1 - d) + src2(i, j).x * d);
			dst(i, j).y = (uchar)(src1(i, j).y * (1 - d) + src2(i, j).y * d);
			dst(i, j).z = (uchar)(src1(i, j).z * (1 - d) + src2(i, j).z * d);
			//printf("HelloWorld! CPU %d %d %d\n", dst(i, j).x, dst(i, j).x, dst(i, j).x);


			//    dst(i, j) = (uchar)(src1(i, j) * (1 - d) + src2(i - Y, j - left)[k] * d);
		}
	}
}

void swap_rb_caller(const PtrStepSz<uchar3>& src, const PtrStepSz<uchar3>& src1, PtrStep<uchar3> dst, int left_x, int right_x, hipStream_t stream)
{
	dim3 block(32, 8);
	dim3 grid((src.cols + block.x - 1) / block.x, (src.rows + block.y - 1) / block.y);

	swap_rb_kernel <<<grid, block, 0, stream>>>(src, src1, dst, left_x, right_x);
	if (stream == 0)
		hipDeviceSynchronize();
}
